#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "cuda_utils.h"
#include "timer.c"

typedef float dtype;

const int T_SIZE = 32;
const int STRIDE = 8;


__global__
void matTrans(dtype* AT, dtype* A, int N)  {
	/* Fill your code here */
  /*
   This is where the actual transpose happens, AT seems like matrix that is the output
   A is the matrix that needs to be transposed, and N is the length.
   The __global__ tells the compiler that this will run on the GPU
  */
  __shared__  dtype scratch[T_SIZE][T_SIZE]; //N should be the size of the tile

  //each thread block should work on a separate tile
  // having a smaller thread count than the tile size is better for some reason
  // Location of x and y is the location on the 2D array
  // Width is the total size of the array
  unsigned int x = blockIdx.x * T_SIZE + threadIdx.x;
  unsigned int y = blockIdx.y * T_SIZE + threadIdx.y;
  unsigned int W = gridDim.x * T_SIZE;

  //stride is the stride size for the rows in the matrix
  // basically this part just copies to local shared mem
  for (int i = 0 ; i < T_SIZE ; i+= STRIDE){
    // from idata is 1 dimensional mapping from a 2D array
      // assert(threadIdx.y + i < T_SIZE);
      // assert(threadIdx.x < T_SIZE);
      // assert((y+i)* width + x < N);
      scratch [threadIdx.y + i][threadIdx.x] = A[(y+i)* W + x];
  }
  __syncthreads();

  x = blockIdx.y * T_SIZE + threadIdx.x;
  y = blockIdx.x * T_SIZE + threadIdx.y;

  for(int i = 0 ; i < T_SIZE; i+= STRIDE){
    // assert(threadIdx.y + i < T_SIZE);
    // assert(threadIdx.x < T_SIZE);
    // assert((y+i)* width + x < N);
    AT[(y + i) * W + x] = scratch[threadIdx.x][threadIdx.y + i];
  }
}

void
parseArg (int argc, char** argv, int* N)
{
	if(argc == 2) {
		*N = atoi (argv[1]);
		assert (*N > 0);
	} else {
		fprintf (stderr, "usage: %s <N>\n", argv[0]);
		exit (EXIT_FAILURE);
	}
}


void
initArr (dtype* in, int N)
{
	int i;

	for(i = 0; i < N; i++) {
		in[i] = (dtype) rand () / RAND_MAX;
	}
}

void
cpuTranspose (dtype* A, dtype* AT, int N)
{
	int i, j;

	for(i = 0; i < N; i++) {
		for(j = 0; j < N; j++) {
			AT[j * N + i] = A[i * N + j];
		}
	}
}

int
cmpArr (dtype* a, dtype* b, int N)
{
	int cnt, i;

	cnt = 0;
	for(i = 0; i < N; i++) {
		if(abs(a[i] - b[i]) > 1e-6) cnt++;
	}

	return cnt;
}



void
gpuTranspose (dtype* A, dtype* AT, int N)
{
  unsigned int s = N * N * sizeof(dtype);
  /*
  This is where the memory needs to be allocated.
  I want to allocate the device memory
  */
  dtype *d_iA;
  dtype *d_oA;
  dim3 gb(N/T_SIZE, N/T_SIZE, 1);
  dim3 tb(T_SIZE, STRIDE, 1);

  CUDA_CHECK_ERROR (hipMalloc (&d_iA, s));
	CUDA_CHECK_ERROR (hipMalloc (&d_oA, s));
  CUDA_CHECK_ERROR (hipMemcpy (d_iA, A, s,hipMemcpyHostToDevice));

  struct stopwatch_t* timer = NULL;
  long double t_gpu;


  /* Setup timers */
  stopwatch_init ();
  timer = stopwatch_create ();
	/* run your kernel here */
  // triple angle brackets mark a call from host to device, launching the kernel
  // launch kernel with this number of blocks,launch kernel with this number of threads
  //warm up
  matTrans <<<gb,tb>>>(d_oA, d_iA,N);
  hipDeviceSynchronize ();
  matTrans <<<gb,tb>>>(d_oA, d_iA,N);
  hipDeviceSynchronize ();
  stopwatch_start (timer);
    matTrans<<<gb,tb>>>(d_oA, d_iA,N);
  hipDeviceSynchronize ();
  t_gpu = stopwatch_stop (timer);
  CUDA_CHECK_ERROR (hipMemcpy (AT, d_oA, s,
				hipMemcpyDeviceToHost));
  fprintf (stdout, "Time to execute sequential index GPU reduction kernel: %Lg secs\n", t_gpu);
  fprintf (stderr, "GPU transpose: %Lg secs ==> %Lg billion elements/second\n",
           t_gpu, (N * N) / t_gpu * 1e-9 );
 double bw = (N * N * sizeof(dtype)) / (t_gpu * 1e9);
 fprintf (stdout, "Effective bandwidth: %.2lf GB/s\n", bw);

}

int
main(int argc, char** argv)
{
  /* variables */
	dtype *A, *ATgpu, *ATcpu;
  int err;

	int N;

  struct stopwatch_t* timer = NULL;
  long double t_cpu;


	N = -1;
	parseArg (argc, argv, &N);

  /* input and output matrices on host */
  /* output */
  ATcpu = (dtype*) malloc (N * N * sizeof (dtype));
  ATgpu = (dtype*) malloc (N * N * sizeof (dtype));

  /* input */
  A = (dtype*) malloc (N * N * sizeof (dtype));

	initArr (A, N * N);

	/* GPU transpose kernel */
  printf("GPU Transpose\n");
	gpuTranspose (A, ATgpu, N);

  /* Setup timers */
  stopwatch_init ();
  timer = stopwatch_create ();

	stopwatch_start (timer);
  /* compute reference array */
	cpuTranspose (A, ATcpu, N);
  t_cpu = stopwatch_stop (timer);
  fprintf (stderr, "Time to execute CPU transpose kernel: %Lg secs\n",
           t_cpu);


  /* check correctness */
	err = cmpArr (ATgpu, ATcpu, N * N);
	if(err) {
		fprintf (stderr, "Transpose failed: %d\n", err);
	} else {
		fprintf (stderr, "Transpose successful\n");
	}

	free (A);
	free (ATgpu);
	free (ATcpu);

  return 0;
}
